#include "hip/hip_runtime.h"
/****************************/
/* THIS IS OPEN SOURCE CODE */
/****************************/

/** 
 * @file    HelloWorld.c
 * @author  Heike Jagode
 *          jagode@eecs.utk.edu
 * Mods:	<your name here>
 *			<your email address>
 * test case for Example component 
 * 
 *
 * @brief
 *  This file is a very simple HelloWorld C example which serves (together
 *	with its Makefile) as a guideline on how to add tests to components.
 *  The papi configure and papi Makefile will take care of the compilation
 *	of the component tests (if all tests are added to a directory named
 *	'tests' in the specific component dir).
 *	See components/README for more details.
 *
 *	The string "Hello World!" is mangled and then restored.
 *
 *  CUDA Context notes for CUPTI_11: Although a hipSetDevice() will create a
 *  primary context for the device that allows kernel execution; PAPI cannot
 *  use a primary context to control the Nvidia Performance Profiler.
 *  Applications must create a context using hipCtxCreate() that will execute
 *  the kernel, this must be done prior to the PAPI_add_events() invocation in
 *  the code below. If multiple GPUs are in use, each requires its own context,
 *  and that context should be active when PAPI_events are added for each
 *  device.  Which means using Seperate PAPI_add_events() for each device. For
 *  an example see simpleMultiGPU.cu.
 * 
 *  There are three points below where hipCtxCreate() is called, this code works
 *  if any one of them is used alone. 
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

#include "papi.h"
#include "papi_test.h"

#define NUM_EVENTS 1
#define PAPI 1
#define STEP_BY_STEP_DEBUG 0 /* helps debug hipCtx_t issues. */

// Prototypes
__global__ void helloWorld(char*);


// Host function
int main(int argc, char** argv)
{
	int retval, quiet = 0;
    hipCtx_t getCtx=NULL, sessionCtx=NULL;
    hipError_t hipError_t;
    hipError_t cuError; (void) cuError;

	/* PAPI Initialization */
	retval = PAPI_library_init( PAPI_VER_CURRENT );
	if( retval != PAPI_VER_CURRENT ) {
		if (!quiet) printf("PAPI init failed\n");
		test_fail(__FILE__,__LINE__,
			"PAPI_library_init failed", 0 );
	}

	if (!quiet) {
		printf( "PAPI_VERSION     : %4d %6d %7d\n",
			PAPI_VERSION_MAJOR( PAPI_VERSION ),
			PAPI_VERSION_MINOR( PAPI_VERSION ),
			PAPI_VERSION_REVISION( PAPI_VERSION ) );
	}
    
	/* Set TESTS_QUIET variable */
	quiet=tests_quiet( argc, argv );
	
#ifdef PAPI
	int i;
	int EventSet = PAPI_NULL;
	long long values[NUM_EVENTS];
	/* REPLACE THE EVENT NAME 'PAPI_FP_OPS' WITH A CUDA EVENT 
	   FOR THE CUDA DEVICE YOU ARE RUNNING ON.
	   RUN papi_native_avail to get a list of CUDA events that are 
	   supported on your machine */
        //char *EventName[] = { "PAPI_FP_OPS" };
        // char const *EventName[] = { "cuda:::fe__cycles_elapsed.sum:device=0"}; // CUPTI_11 event. 
        char const *EventName[] = { "cuda:::dram__bytes_read.sum:device=0"}; // CUPTI_11 event.
        // 2 pass var. char const *EventName[] = { "cuda:::dram__bytes.avg.pct_of_peak_burst_elapsed:device=0"};
	int events[NUM_EVENTS];
	int eventCount = 0;

	/* convert PAPI native events to PAPI code */
	for( i = 0; i < NUM_EVENTS; i++ ){
                retval = PAPI_event_name_to_code( (char *)EventName[i], &events[i] );
		if( retval != PAPI_OK ) {
			fprintf(stderr, "%s:%s:%i PAPI_event_name_to_code failed for '%s'\n", __FILE__, __func__, __LINE__, EventName[i] );
			continue;
		}
		eventCount++;
		if (!quiet) printf( "Name %s --- Code: %#x\n", EventName[i], events[i] );
	}

	/* if we did not find any valid events, just report test failed. */
	if (eventCount == 0) {
		if (!quiet) printf( "Test FAILED: no valid events found.\n");
		test_skip(__FILE__,__LINE__,"No events found",0);
		return 1;
	}
	
    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i before PAPI_create_eventset() getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

	retval = PAPI_create_eventset( &EventSet );
	if( retval != PAPI_OK ) {
		if (!quiet) printf( "PAPI_create_eventset failed\n" );
		test_fail(__FILE__,__LINE__,"Cannot create eventset",retval);
	}	

    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }
        
    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i before PAPI_add_events(), getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

    // If multiple GPUs/contexts were being used, you'd need to
    // create contexts for each device. See, for example,
    // simpleMultiGPU.cu.

    // Context Create. We will use this one to run our kernel.
    hipCtxCreate(&sessionCtx, 0, 0); // Create a context, NULL flags, Device 0.

    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after hipCtxCreate(&sessionCtx), about to PAPI_start(), sessionCtx=%p, getCtx=%p.\n", __FILE__, __func__, __LINE__, sessionCtx, getCtx);
    }

    retval = PAPI_add_events( EventSet, events, eventCount );
	if( retval != PAPI_OK ) {
		fprintf( stderr, "PAPI_add_events failed\n" );
	}

    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i before PAPI_start(), getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

	retval = PAPI_start( EventSet );
	if( retval != PAPI_OK ) {
		fprintf( stderr, "PAPI_start failed\n" );
	}

    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after PAPI_start(), getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

#endif

	int j;
	
	// desired output
	char str[] = "Hello World!";

	// mangle contents of output
	// the null character is left intact for simplicity
	for(j = 0; j < 12; j++) {
		str[j] -= j;
	}

    printf("mangled str=%s\n", str);

	// allocate memory on the device
	char *d_str;
	size_t size = sizeof(str);
	hipMalloc((void**)&d_str, size);
	
    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after hipMalloc() getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

	// copy the string to the device
	hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);
	
    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after hipMemcpy(ToDevice) getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

	// set the grid and block sizes
	dim3 dimGrid(2); // one block per word
	dim3 dimBlock(6); // one thread per character

	// invoke the kernel
	helloWorld<<< dimGrid, dimBlock >>>(d_str);

    hipError_t = hipGetLastError();
    if (STEP_BY_STEP_DEBUG) {
        fprintf(stderr, "%s:%s:%i Kernel Return Code: %s.\n", __FILE__, __func__, __LINE__, hipGetErrorString(hipError_t));
    }

    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i After Kernel Execution: getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

	// retrieve the results from the device
	hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);
	
    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after hipMemcpy(ToHost) getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

	// free up the allocated memory on the device
	hipFree(d_str);
	
    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after hipFree() getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

	if (!quiet) printf("END: %s\n", str);

	
#ifdef PAPI
	retval = PAPI_read( EventSet, values );
	if( retval != PAPI_OK )
		fprintf(stderr, "PAPI_read failed, ret=%d (%s)\n", retval, PAPI_strerror(retval) );

    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after PAPI_read getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

	for( i = 0; i < eventCount; i++ )
		if (!quiet) printf( "read: %12lld \t=0X%016llX \t\t --> %s \n", values[i], values[i], EventName[i] );

    retval = hipCtxPopCurrent(&getCtx);
	if( retval != hipSuccess)
		fprintf( stderr, "hipCtxPopCurrent failed, retval=%d (%s)\n", retval, PAPI_strerror(retval) );
    
    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after hipCtxPopCurrent() getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }
   
	retval = PAPI_stop( EventSet, values );
	if( retval != PAPI_OK )
		fprintf( stderr, "PAPI_stop failed, retval=%d (%s)\n", retval, PAPI_strerror(retval) );

    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after PAPI_stop getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

	retval = PAPI_cleanup_eventset(EventSet);
	if( retval != PAPI_OK )
		fprintf(stderr, "PAPI_cleanup_eventset failed, retval=%d (%s)\n", retval, PAPI_strerror(retval) );

    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after PAPI_cleanup_eventset getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

	retval = PAPI_destroy_eventset(&EventSet);
	if (retval != PAPI_OK)
		fprintf(stderr, "PAPI_destroy_eventset failed, retval=%d (%s)\n", retval, PAPI_strerror(retval) );

    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after PAPI_destroy_eventset getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }


	for( i = 0; i < eventCount; i++ )
		if (!quiet) printf( "stop: %12lld \t=0X%016llX \t\t --> %s \n", values[i], values[i], EventName[i] );
#endif

    if (STEP_BY_STEP_DEBUG) {
        fprintf(stderr, "%s:%s:%i before hipCtxDestroy sessionCtx=%p.\n", __FILE__, __func__, __LINE__, sessionCtx);
    }

    // Test destroying the session Context.
    if (sessionCtx != NULL) {
        hipCtxDestroy(sessionCtx);
    }

    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after hipCtxDestroy(%p) getCtx=%p.\n", __FILE__, __func__, __LINE__, sessionCtx, getCtx);
    }

	PAPI_shutdown();

    if (STEP_BY_STEP_DEBUG) {
        hipCtxGetCurrent(&getCtx);
        fprintf(stderr, "%s:%s:%i after PAPI_shutdown getCtx=%p.\n", __FILE__, __func__, __LINE__, getCtx);
    }

	test_pass(__FILE__);

	return 0;
}


// Device kernel
__global__ void
helloWorld(char* str)
{
	// determine where in the thread grid we are
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// unmangle output
	str[idx] += idx;
}

